#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void kernel()
{
	int tid = blockIdx.x;
	printf("Block number %d\n", tid);
}

int main(void)
{
	int* dev_a;
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel << <10, 1 >> > ();
	hipFree(dev_a);
	return 0;
}