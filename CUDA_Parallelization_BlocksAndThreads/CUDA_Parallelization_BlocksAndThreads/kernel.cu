#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
const int ThreadCount=15;
const int BlockCount=10;

__global__ void kernel()
{
    int tid_block = blockIdx.x;
    int tid_thread = threadIdx.x;
    printf("Block number %d. Thread number %d\n",tid_block, tid_thread);
}
int main(void)
{
	int* dev_a;
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel << <BlockCount, ThreadCount>> > ();
	hipFree(dev_a);
	return 0;
}