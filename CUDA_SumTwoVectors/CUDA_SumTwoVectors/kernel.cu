#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
const int N = 10;

__global__ void sumTwoVectors(int* a, int* b, int* c)
{
	int tid = blockIdx.x;
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		printf("Blok nömrəsi %d write %d massiv %d mövqe\n", tid, c[tid], tid);
	}
}

int main()
{
	int host_a[N], host_b[N], host_c[N];
	int* dev_a, * dev_b, * dev_c;
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	for (int i = 0; i < N; i++)
	{
		host_a[i] = rand()%100;
		host_b[i] = rand()%100;
	}
	hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);
	sumTwoVectors << <N, 1 >> > (dev_a, dev_b, dev_c);
	hipMemcpy(host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}