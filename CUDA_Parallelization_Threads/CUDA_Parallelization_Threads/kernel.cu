#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void kernel()
{	
	int tid = threadIdx.x;
	printf("Thread number %d\n", tid);
}

int main(void)
{
	int* dev_a;
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel << <1, 10 >> > ();
	hipFree(dev_a);
	return 0;
}