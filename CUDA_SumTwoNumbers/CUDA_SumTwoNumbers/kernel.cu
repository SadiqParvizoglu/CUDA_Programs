#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void sumTwoNumbers(int a,int b,int *c)
{
    *c = a + b;
}

int main()
{
    int host_c;
    int* dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    sumTwoNumbers << <1, 1 >> > (17, 24, dev_c);
    hipMemcpy(&host_c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
    printf("17 + 24 = %d\n", host_c);
    hipFree(dev_c);
    return 0;
}