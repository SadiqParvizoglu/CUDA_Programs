#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void HelloWorld()
{
    printf("Hello World, %d, %d\n",blockIdx.x,threadIdx.x);
}

int main()
{
    HelloWorld << <1, 1 >> > ();
    return 0;
}